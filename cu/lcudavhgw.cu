#include "hip/hip_runtime.h"
#include <npp.h>

#define BORDER_VALUE 255

#define PRINT_ON

#ifndef PRINTF
# ifndef PRINT_ON
#  define PRINTF(...) ((void)0)
# else
#  define PRINTF(fmt,...) (printf(fmt, ## __VA_ARGS__))
# endif
#endif



template <class dataType, morphOperation MOP>
__global__ void _verticalVHGWKernel(const dataType *img, int imgStep, dataType *result,
                                    int resultStep, unsigned int width, unsigned int height,
                                        unsigned int size, NppiSize borderSize) {
    const unsigned int y      = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	  const unsigned int step   = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    const unsigned int startx = __umul24(step,size);

    if (y >= height || startx > width)
        return;

    const dataType *lineIn = img+y;
    dataType *lineOut      = result+y;

    const unsigned int center  = startx + (size-1);

    dataType minarray[512];
    minarray[size-1] = lineIn[center*imgStep];

    dataType nextMin;
    unsigned int k;
    if (MOP == ERODE) {
        for(k=1;k<size; ++k) {
            nextMin = lineIn[(center-k)*imgStep];
            minarray[size-1-k] = min(minarray[size-k], nextMin);

            nextMin = (center+k < height+size-1) ? lineIn[(center+k)*imgStep] : 255;
            minarray[size-1+k] = min(minarray[size+k-2], nextMin);
        }
    } else {
        for(k=1;k<size; ++k) {
            nextMin = lineIn[__umul24(center-k,imgStep)];
            minarray[size-1-k] = max(minarray[size-k], nextMin);

            nextMin = lineIn[__umul24(center+k,imgStep)];
            minarray[size-1+k] = max(minarray[size+k-2], nextMin);
        }
    }

    int diff = height - startx;
    if (diff > 0) {
        lineOut += startx*resultStep;
        lineOut[0] = minarray[0];

        for(k=1; k < size-1; ++k) {
            if (diff > k) {
                lineOut[k*resultStep] = minMax<dataType, MOP>(minarray[k], minarray[k+size-1]);
            }
        }

        if (diff > size-1) {
            lineOut[(size-1)*resultStep] = minarray[2*(size-1)];
        }
    }
}

#if 0

template <class dataType, morphOperation MOP>
__global__ void _horizontalVHGWKernel(const dataType *img, int imgStep, dataType *result,
                                    int resultStep, unsigned int width, unsigned int height,
                                        unsigned int size, NppiSize borderSize) {
    const unsigned int x      = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    const unsigned int step   = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    const unsigned int starty = __umul24(step,size);

    if (x >= width || starty > height)
        return;

    const dataType *lineIn = img+x*imgStep;
    dataType *lineOut      = result+x*resultStep;
    const unsigned int center  = starty + (size-1);

    dataType minarray[512];
    minarray[size-1] = lineIn[center];

    dataType nextMin;
    unsigned int k;
    if (MOP == ERODE) {
        for(k=1;k<size; ++k) {
            nextMin = lineIn[center-k];
            minarray[size-1-k] = min(minarray[size-k], nextMin);

            nextMin = (center+k < width+size-1) ? lineIn[center+k] : BORDER_VALUE;
            minarray[size-1+k] = min(minarray[size+k-2], nextMin);
        }
    } else {
        for(k=1;k<size; ++k) {
            nextMin = lineIn[center-k];
            minarray[size-1-k] = max(minarray[size-k], nextMin);

            nextMin =lineIn[center+k];
            minarray[size-1+k] = max(minarray[size+k-2], nextMin);
        }
    }

    int diff = width - starty;
    if (diff > 0) {
        lineOut += starty;
        lineOut[0] = minarray[0];

        for(k=1; k < size-1; ++k) {
            if (diff > k) {
                lineOut[k] = minMax<dataType, MOP>(minarray[k], minarray[k+size-1]);
            }
        }

        if (diff > size-1) {
            lineOut[size-1] = minarray[2*(size-1)];
        }
    }
}

#else
# if 1
template <class dataType, morphOperation MOP>
__global__ void _horizontalVHGWKernel(const dataType *img, int imgStep,
                                      dataType *result, int resultStep,
                                      unsigned int width, unsigned int height,
                                      unsigned int size, NppiSize borderSize)
{
    #define LINEC 13
    #define LINES 1040
    __shared__ dataType imHx[LINEC * LINES];
    __shared__ dataType imGx[LINEC * LINES];
    dataType *imHxPtr, *imGxPtr;
    dataType *imHxStepPtr, *imGxStepPtr;
    uint32_t ptroffset;

    dataType localSrc[13];
    uint32_t startx = __umul24(size, threadIdx.x);
    uint32_t imline = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    dataType *dstptr;
    const dataType *srcptr;
    char pred = !(imline >= height) && !((startx - size) >= width);

    //Load data from global memory to shared memory
    ptroffset = threadIdx.y * LINES;
    imGxPtr = imGx + ptroffset;
    imHxPtr = imHx + ptroffset;
    srcptr = img + imline * imgStep + startx;
    imGxStepPtr = imGxPtr + startx;
    imHxStepPtr = imHxPtr + startx;


    printf("111\n");
    if (pred) {
      asm("prefetch.global.L1 [%0];"::"r"(srcptr));
      localSrc[0] = srcptr[0];
      localSrc[1] = srcptr[1];
      localSrc[2] = srcptr[2];
      localSrc[3] = srcptr[3];
      localSrc[4] = srcptr[4];
      localSrc[5] = srcptr[5];
      localSrc[6] = srcptr[6];
      localSrc[7] = srcptr[7];
      localSrc[8] = srcptr[8];
      localSrc[9] = srcptr[9];
      localSrc[10] = srcptr[10];
      localSrc[11] = srcptr[11];
      localSrc[12] = srcptr[12];

      //Processing
      dataType gxMax, hxMax;
      imGxStepPtr[0] = gxMax = localSrc[0];
      imGxStepPtr[1] = gxMax = max(gxMax, localSrc[1]);
      imGxStepPtr[2] = gxMax = max(gxMax, localSrc[2]);
      imGxStepPtr[3] = gxMax = max(gxMax, localSrc[3]);
      imGxStepPtr[4] = gxMax = max(gxMax, localSrc[4]);
      imGxStepPtr[5] = gxMax = max(gxMax, localSrc[5]);
      imGxStepPtr[6] = gxMax = max(gxMax, localSrc[6]);
      imGxStepPtr[7] = gxMax = max(gxMax, localSrc[7]);
      imGxStepPtr[8] = gxMax = max(gxMax, localSrc[8]);
      imGxStepPtr[9] = gxMax = max(gxMax, localSrc[9]);
      imGxStepPtr[10] = gxMax = max(gxMax, localSrc[10]);
      imGxStepPtr[11] = gxMax = max(gxMax, localSrc[11]);
      imGxStepPtr[12] = gxMax = max(gxMax, localSrc[12]);

      imHxStepPtr[12] = hxMax = localSrc[12];
      imHxStepPtr[11] = hxMax = max(hxMax, localSrc[11]);
      imHxStepPtr[10] = hxMax = max(hxMax, localSrc[10]);
      imHxStepPtr[9] = hxMax = max(hxMax, localSrc[9]);
      imHxStepPtr[8] = hxMax = max(hxMax, localSrc[8]);
      imHxStepPtr[7] = hxMax = max(hxMax, localSrc[7]);
      imHxStepPtr[6] = hxMax = max(hxMax, localSrc[6]);
      imHxStepPtr[5] = hxMax = max(hxMax, localSrc[5]);
      imHxStepPtr[4] = hxMax = max(hxMax, localSrc[4]);
      imHxStepPtr[3] = hxMax = max(hxMax, localSrc[3]);
      imHxStepPtr[2] = hxMax = max(hxMax, localSrc[2]);
      imHxStepPtr[1] = hxMax = max(hxMax, localSrc[1]);
      imHxStepPtr[0] = hxMax = max(hxMax, localSrc[0]);
    }

    __syncthreads();
    if(pred) {
      //Save data fromshared memory to global memory
      imHxStepPtr -= 6;
      imGxStepPtr += 6;
      dstptr = result + imline * resultStep + startx;
      dstptr[0] = max(imGxStepPtr[0], imHxStepPtr[0]);
      dstptr[1] = max(imGxStepPtr[1], imHxStepPtr[1]);
      dstptr[2] = max(imGxStepPtr[2], imHxStepPtr[2]);
      dstptr[3] = max(imGxStepPtr[3], imHxStepPtr[3]);
      dstptr[4] = max(imGxStepPtr[4], imHxStepPtr[4]);
      dstptr[5] = max(imGxStepPtr[5], imHxStepPtr[5]);
      dstptr[6] = max(imGxStepPtr[6], imHxStepPtr[6]);
      dstptr[7] = max(imGxStepPtr[7], imHxStepPtr[7]);
      dstptr[8] = max(imGxStepPtr[8], imHxStepPtr[8]);
      dstptr[9] = max(imGxStepPtr[9], imHxStepPtr[9]);
      dstptr[10] = max(imGxStepPtr[10], imHxStepPtr[10]);
      dstptr[11] = max(imGxStepPtr[11], imHxStepPtr[11]);
      dstptr[12] = max(imGxStepPtr[12], imHxStepPtr[12]);
    }
}
# else
template <class dataType, morphOperation MOP>
__global__ void _horizontalVHGWKernel(const dataType *img, int imgStep,
                                      dataType *result, int resultStep,
                                      unsigned int width, unsigned int height,
                                      unsigned int size, NppiSize borderSize)
{
    #define LINEC 13
    #define LINES 1040
    __shared__ dataType imHx[LINEC * LINES];
    __shared__ dataType imGx[LINEC * LINES];
    dataType *imHxPtr, *imGxPtr;
    dataType *imHxStepPtr, *imGxStepPtr;
    uint32_t ptroffset;
    uint32_t j;

    dataType localSrc[13];
    uint32_t startx = __umul24(size, threadIdx.x);
    uint32_t imline = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    dataType *dstptr;
    const dataType *srcptr;
    char pred = !(imline >= height) && !((startx - size) >= width);

    //Load data from global memory to shared memory
    ptroffset = threadIdx.y * LINES;
    imGxPtr = imGx + ptroffset;
    imHxPtr = imHx + ptroffset;
    srcptr = img + imline * imgStep + startx;
    imGxStepPtr = imGxPtr + startx;
    imHxStepPtr = imHxPtr + startx;

    if (pred) {
      asm("prefetch.global.L1 [%0];"::"r"(srcptr));
      for (int i = 0; i < size; i++) localSrc[i] = srcptr[i];
      //Processing
      dataType gxMax, hxMax;

      imGxStepPtr[0] = gxMax = localSrc[0];
      for (int i = 1; i < size; i++) imGxStepPtr[i] = gxMax = max(gxMax, localSrc[i]);

      imHxStepPtr[12] = hxMax = localSrc[12];
      for (int i = 11; i >= 0; i--) imHxStepPtr[i] = hxMax = max(hxMax, localSrc[i]);
    }

    __syncthreads();
    if(pred) {
      //Save data fromshared memory to global memory
      imHxStepPtr -= 6;
      imGxStepPtr += 6;
      dstptr = result + imline * resultStep + startx;
      j = 12;
      do {*(dstptr++) = max(*(imGxStepPtr++), *(imHxStepPtr++)); } while(j--);
    }
}

# endif
#endif

/*{
    dataType minarray[512];
    dataType *inputRow, *lineOut;

	const unsigned int y    = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	const unsigned int step = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;

    const unsigned int startx = __umul24(step,size);
    if (y >= height + size/2 || startx > width)
        return;

    inputRow = (dataType*)img + y*imgStep;
    lineOut = result + y*resultStep;

    const unsigned int windowCenter  = step*size+(size-1);
    unsigned int k;

    minarray[size-1] = inputRow[windowCenter];
    dataType nextMin;

    if (MOP == ERODE) {
        for(k=1;k<size; ++k) {
            nextMin = inputRow[windowCenter-k];
            minarray[size-1-k] = min(minarray[size-k], nextMin);

            nextMin = inputRow[windowCenter+k];
            minarray[size-1+k] = min(minarray[size+k-2], nextMin);
        }
    } else {
        for(k=1;k<size; ++k) {
            nextMin = inputRow[windowCenter-k];
            minarray[size-1-k] = max(minarray[size-k], nextMin);

            nextMin = inputRow[windowCenter+k];
            minarray[size-1+k] = max(minarray[size+k-2], nextMin);
        }
    }

    int hdiff = height - startx;
    if (0 < hdiff) {
        lineOut += startx;

        lineOut[0] = minarray[0];

        for(k=1; k < size-1; ++k) {
            if (k <= hdiff) {
                lineOut[k] = minMax<dataType, MOP>(minarray[k], minarray[k+size-1]);
            }
        }

        if (size-1 <= hdiff) {
            lineOut[size-1] = minarray[__umul24(2,size-1)];
        }
    }
}*/


template <class dataType, morphOperation MOP, vhgwDirection DIRECTION>
NppStatus _globalVHGW(const dataType * img, Npp32s imgStep, dataType * result,
                        Npp32s resultStep, NppiSize oSizeROI, unsigned int size,
                            NppiSize borderSize) {
    const unsigned int width = oSizeROI.width;
    const unsigned int height = oSizeROI.height;

    PRINTF("width %d, height %d\n", width, height);
    PRINTF("Border (w: %d , h: %d)\n", borderSize.width, borderSize.height);

    unsigned int steps;
    if (DIRECTION == VERTICAL) {
        steps = (width+size-1)/size;
        dim3 gridSize((steps+128-1)/128, (height+2-1)/2);
        dim3 blockSize(128,2);

        _verticalVHGWKernel<dataType, MOP><<<gridSize,blockSize>>>
            (img, imgStep,result, resultStep, width, height, size, borderSize);
    }
    else { // HORIZONTAL
        int linesblock;
        int lines;
        dim3 gridSize;
        dim3 blockSize;

        steps = width / size;

        lines = 16384 / width;
        if (lines * steps > 1024) lines = 1024 / steps;

        linesblock = (height % lines) ? height / lines + 1 : height / lines;

        blockSize = dim3(steps, lines);
        gridSize = dim3(1, linesblock);
        printf("Block size (%d,%d)\n", steps, lines);
        printf("Grid size (%d,%d)\n", 1, linesblock);
#if 0
        _horizontalVHGWKernel<dataType, MOP><<<gridSize,blockSize>>>
            (img, imgStep,result, resultStep, width, height, size, borderSize);
#else
        hipModule_t module;
        hipFunction_t function;
        hipError_t err;

        const char* module_file = "horizontal13.ptx";
        const char* kernel_name = "vhgw_horizontal13";
        const char *errstr;

        printf("Loading ptx!\n");
        err = hipModuleLoad(&module, module_file);
        if (hipSuccess != err) {
          printf("Failed to load module\n");
          hipDrvGetErrorString(err, &errstr);
          printf("%s\n", errstr);
          exit(255);
        }

        printf("Loading function!\n");
        err = hipModuleGetFunction(&function, module, kernel_name);
        if (hipSuccess != err) {
          printf("Failed to load function\n");
          hipDrvGetErrorString(err, &errstr);
          printf("%s\n", errstr);
          exit(255);
        }

        printf("Launching kernel!\n");

        /*
           hipError_t hipModuleLaunchKernel (
           hipFunction_t f,
           unsigned int  gridDimX, unsigned int  gridDimY, unsigned int  gridDimZ,
           unsigned int  blockDimX, unsigned int  blockDimY, unsigned int  blockDimZ,
           unsigned int  sharedMemBytes,
           hipStream_t hStream,
           void** kernelParams,
           void** extra )
        */

        cuParamSetSize(function, 7 * 4);
        cuParamSetv(function, 0, (void *)&img, 4);
        cuParamSetv(function, 4, (void *)&imgStep, 4);
        cuParamSetv(function, 8, (void *)&result, 4);
        cuParamSetv(function, 12, (void *)&resultStep, 4);
        cuParamSetv(function, 16, (void *)&width, 4);
        cuParamSetv(function, 20, (void *)&height, 4);
        cuParamSetv(function, 24, (void *)&size, 4);
        cuFuncSetBlockShape (function, steps, lines, 1);
        cuFuncSetSharedSize (function, 24000);
        cuLaunchGrid (function, 1, linesblock);

        if (hipSuccess != err) {
          printf("Failed to launch function\n");
          hipDrvGetErrorString(err, &errstr);
          printf("%d: %s\n", err, errstr);
          exit(255);
        }
        hipDeviceSynchronize();
#endif
    }

    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
       //print the CUDA error message and exit
       PRINTF("CUDA error: %s\n", hipGetErrorString(error));
       exit(-1);
    }

    return NPP_SUCCESS;
}

/*
    Function for writing images in .PGM format. Useful for debugging, to track image changes step by step.

    void writeImageToPGM(const char* filename, const unsigned char* dev, int devStep, unsigned int width, unsigned int height) {
    int r,c;
    unsigned char *host = (unsigned char*)malloc(width*height);

    hipMemcpy2D((void*)host, width, dev, devStep, width, height, hipMemcpyDeviceToHost);

    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA writeImageToPGM error: %s\n", hipGetErrorString(error));
       // exit(-1);
    } else {
        FILE *file;
        file = fopen(filename, "w");
        fprintf(file,"P5\n%d %d\n255\n", height, width);
        for(c = 0; c < width; c++) {
        	for(r = 0; r < height; r++) {
	            fputc(host[r*width + c],file);
	        }
        }
        fclose(file);
    }
}*/
